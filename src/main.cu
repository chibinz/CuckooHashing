#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

#include "device.h"
#include "host.h"
#include "multi.h"

int main(int argc, char **argv) {
  if (argc != 3) {
    printf("Usage: %s <width> <load-factor>\n", argv[0]);
    return -1;
  }

  auto width = atoi(argv[1]);
  auto load = atof(argv[2]);
  auto entry = (u32)((1 << width) * load);
  // auto t = new DeviceTable(1 << width, entry);
  auto t = new MultilevelTable(1 << width, entry);

  u32 *array, *set;
  hipMallocManaged(&array, sizeof(u32) * entry);
  hipMallocManaged(&set, sizeof(u32) * entry);
  hipMemset(set, 0, sizeof(u32) * entry);
  randomizeDevice(array, entry);
  syncCheck();

  t->insert(array);
  t->lookup(array, set);

  syncCheck();
  syncCheck();

  // printf("Total number of collisions: %u\n", t->collision);
  syncCheck();

  hipFree(array);
  delete t;

  return 0;
}
