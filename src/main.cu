#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

#include "bench.h"
#include "device.h"
#include "host.h"
#include "multi.h"

int main(int argc, char **argv) {
  if (argc != 3) {
    printf("Usage: %s <width> <load-factor>\n", argv[0]);
    return -1;
  }

  auto width = atoi(argv[1]);
  auto load = atof(argv[2]);
  auto entry = (u32)((1 << width) * load);
  // auto t = new DeviceTable(1 << width);
  auto t = new MultilevelTable(1 << width);

  insertion();

  u32 *key, *set;
  hipMalloc(&key, sizeof(u32) * entry);
  hipMalloc(&set, sizeof(u32) * entry);
  hipMemset(set, 0, sizeof(u32) * entry);
  randomizeDevice(key, entry);
  syncCheck();

  t->insert(key, entry);
  t->lookup(key, set, entry);

  syncCheck();

  hipFree(key);
  delete t;

  return 0;
}
