#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

#include "DeviceTable.h"
#include "HostTable.h"
#include "Types.h"
#include "xxHash.h"

int main() {
  auto t = HostTable(10, 2);

  for (usize i = 0; i < 10; i += 1) {
    t.insert(rand());
    t.write(stdout);
    putchar('\n');
  }

  hipDeviceSynchronize();

  wrapper();

  auto err = hipGetLastError(); // Get error code

  if (err != hipSuccess) {
    printf("Error: %s!\n", hipGetErrorString(err));
    return -1;
  }

  hipDeviceSynchronize();

  return 0;
}
