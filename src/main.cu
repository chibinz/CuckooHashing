#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

#include "DeviceTable.h"
#include "HostTable.h"
#include "MultilevelTable.h"

int main(int argc, char **argv) {
  if (argc != 3) {
    printf("Usage: %s <width> <load-factor>\n", argv[0]);
    return -1;
  }

  auto width = atoi(argv[1]);
  auto load = atof(argv[2]);
  auto entry = (u32)((1 << width) * load);
  // auto t = new DeviceTable(1 << width, entry);
  auto t = new MultilevelTable(1 << width, entry);

  u32 *array, *set;
  hipMalloc(&array, sizeof(u32) * entry);
  hipMallocManaged(&set, sizeof(u32) * entry);
  hipMemset(set, 0, sizeof(u32) * entry);
  randomizeDevice(array, entry);
  syncCheck();

  t->insert(array);
  t->lookup(array, set);

  for (u32 i = 0; i < entry; i++) {
    printf("%x\n", set[i]);
  }

  printf("Total number of collisions: %u\n", t->collision);
  syncCheck();

  hipFree(array);
  delete t;

  return 0;
}
