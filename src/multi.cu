#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include "common.h"
#include "multi.h"
#include "types.h"
#include "xxhash.h"

namespace {

__global__ void divideKernel(MultilevelTable *t, u32 *array, u32 n) {
  u32 id = threadIdx.x + blockDim.x * blockIdx.x;

  if (id < n) {
    u32 b = xxhash(t->bucketSeed, array[id]) % t->bucket;
    u32 old = atomicAdd(&t->bucketSize[b], 1);
    if (old < t->bucketCapacity) {
      t->bucketData[b * t->bucketCapacity + old] = array[id];
    } else {
      printf("Bucket overflow! %u\n", b);
      atomicAdd(&t->collision, 1);
    }
  }
}

__global__ void insertKernel(MultilevelTable *t) {
  // Declare shared memory size of `t->dim * t->len`
  extern __shared__ u32 local[];

  // Initialize shared memory
  for (u32 i = threadIdx.x; i < t->dim * t->len; i += blockDim.x)
    local[i] = empty;
  __syncthreads();

  u32 bid = blockIdx.x;
  u32 tid = threadIdx.x;

  if (tid < t->bucketSize[bid]) {
    u32 k = t->bucketData[bid * t->bucketCapacity + tid];
    // printf("%d\n", k);

    do {
      // Record collision in shared memory
      local[t->dim * t->len] = 0;

      for (u32 i = 0; i < t->threshold && k != empty; i += 1) {
        u32 d = i % t->dim;
        u32 key = xxhash(t->seed[bid * t->dim + d], k) % t->len;
        // k = atomicExch(&local[d * t->len + key], k);
        k = atomicExch(&t->val[bid * t->len * t->dim + d * t->len + key], k);
      }

      // Guard to avoid bank conflict
      if (local[t->dim * t->len] == 0 && k != empty) {
        local[t->dim * t->len] = 1;
        // for (u32 d = 0; d < t->dim; d += 1) {
        //   t->seed[bid * t->dim + d] = xxhash(tid, t->seed[bid * t->dim + d]);
        // }
      }
      __syncthreads();

    } while (local[t->dim * t->len] != 0);
  }

  // Copy value from shared memory to global memory
  for (u32 i = threadIdx.x; i < t->dim * t->len; i += blockDim.x) {
    // t->val[bid * t->len * t->dim + i] = local[i];
    // printf("%d\n", local[i]);
  }
}

__global__ void lookupKernel(MultilevelTable *t, u32 *keys, u32 *set, u32 n) {
  u32 id = threadIdx.x + blockDim.x * blockIdx.x;

  if (id < n) {
    u32 k = keys[id];
    u32 b = xxhash(t->bucketSeed, keys[id]) % t->bucket;

    for (u32 d = 0; d < t->dim; d += 1) {
      u32 key = xxhash(t->seed[b * t->dim + d], k) % t->len;
      if (k == t->val[b * t->len * t->dim + d * t->len + key]) {
        set[id] = 1;
      }
    }
  }
}

} // namespace

MultilevelTable::MultilevelTable(u32 capacity, u32 entry) {
  dim = 3;
  len = 192;
  size = entry;
  collision = 0;
  bucketCapacity = 512;
  bucket = ceil(capacity, bucketCapacity);
  thread = bucketCapacity;
  block = bucket;
  threshold = 4 * bit_width(dim * len);

  hipMallocManaged(&val, sizeof(u32) * dim * len * bucket);
  hipMallocManaged(&seed, sizeof(u32) * dim * bucket);
  hipMallocManaged(&bucketSize, sizeof(u32) * bucket);
  hipMallocManaged(&bucketData, sizeof(u32) * bucketCapacity * bucket);

  hipMemset(val, -1, sizeof(u32) * dim * len * bucket);
  hipMemset(bucketSize, 0, sizeof(u32) * bucket);
  randomizeDevice(seed, dim * bucket);
}

MultilevelTable::~MultilevelTable() {
  hipFree(bucketSize);
  hipFree(bucketData);
}

void MultilevelTable::insert(u32 *k) {
  do {
    collision = 0;
    bucketSeed = rand();
    hipMemset(bucketSize, 0, sizeof(u32) * bucket);
    divideKernel<<<block, thread>>>(this, k, size);
    syncCheck();
  } while (collision > 0);

  insertKernel<<<block, thread, sizeof(u32) * (dim * len + 1)>>>(this);
  syncCheck();
}

void MultilevelTable::lookup(u32 *k, u32 *s) {
  lookupKernel<<<block, thread>>>(this, k, s, size);
  syncCheck();
}
