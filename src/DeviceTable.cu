#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Common.h"
#include "DeviceTable.h"
#include "Types.h"
#include "xxHash.h"

__global__ void randomizeArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 acc = xxhash(array[id], n);
    acc = xxhash(acc, threadIdx.x);
    acc = xxhash(acc, blockIdx.x);
    acc = xxhash(acc, blockDim.x);
    array[id] = (u32)(acc);
  }
}

__global__ void printArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    printf("%08x: %x\n", id, array[id]);
  }
}

__global__ void setEmpty(u32 *val, u32 capacity) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < capacity) {
    val[id] = empty;
  }
}

__global__ void batchedInsert(DeviceTable *t, u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = array[id];

    for (u32 i = 0; i < t->threshold && v != empty; i += 1) {
      u32 b = i % t->dim;
      u32 key = xxhash(t->seed[b], v) % t->len;
      v = atomicExch(&t->val[b * t->len + key], v);
    }

    // Record number of collisions
    if (v != empty) {
      atomicAdd(&t->collision, 1);
    }
  }
}

__global__ void batchedLookup(DeviceTable *t, u32 *keys, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = keys[id];

    for (u32 i = 0; i < t->dim; i += 1) {
      u32 key = xxhash(t->seed[i], v) % t->len;
      if (t->val[i * t->len + key] == v) {
        break;
      }
    }
  }
}

void randomizeGPU(u32 *array, u32 n) {
  randomizeArray<<<n / 256 + 1, 256>>>(array, n);
}

void DeviceTable::insert(u32 *v) {
  u32 numEntries = 1 << 24;
  u32 numThreads = 1024;
  u32 entryBlocks = numEntries / numThreads;

  batchedInsert<<<entryBlocks, numThreads>>>(this, v, numEntries);
  syncCheck();
  while (collision > 0) {
    reset();
    batchedInsert<<<entryBlocks, numThreads>>>(this, v, numEntries);
    syncCheck();
  }
}

void DeviceTable::lookup(u32 *k) {
  u32 numEntries = 1 << 24;
  u32 numThreads = 1024;
  u32 entryBlocks = numEntries / numThreads;
  batchedLookup<<<entryBlocks, numThreads>>>(this, k, numEntries);
  syncCheck();
}
