#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Common.h"
#include "DeviceTable.h"
#include "Types.h"
#include "xxHash.h"

namespace {

__global__ void randomizeKernel(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 acc = xxhash(array[id], n);
    acc = xxhash(acc, threadIdx.x);
    acc = xxhash(acc, blockIdx.x);
    acc = xxhash(acc, blockDim.x);
    array[id] = (u32)(acc);
  }
}

__global__ void insertKernel(DeviceTable *t, u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = array[id];

    for (u32 i = 0; i < t->threshold && v != empty; i += 1) {
      u32 b = i % t->dim;
      u32 key = xxhash(t->seed[b], v) % t->len;
      v = atomicExch(&t->val[b * t->len + key], v);
    }

    // Record number of collisions
    if (v != empty) {
      atomicAdd(&t->collision, 1);
    }
  }
}

__global__ void lookupKernel(DeviceTable *t, u32 *keys, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = keys[id];

    for (u32 i = 0; i < t->dim; i += 1) {
      u32 key = xxhash(t->seed[i], v) % t->len;
      if (t->val[i * t->len + key] == v) {
        break;
      }
    }
  }
}

} // namespace

void randomizeDevice(u32 *array, u32 n) {
  randomizeKernel<<<n / 256 + 1, 256>>>(array, n);
}

void DeviceTable::insert(u32 *v) {
  do {
    reset();
    insertKernel<<<block, thread>>>(this, v, size);
    syncCheck();
  } while (collision > 0);
}

void DeviceTable::lookup(u32 *k) {
  lookupKernel<<<block, thread>>>(this, k, size);
  syncCheck();
}
