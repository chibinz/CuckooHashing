#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Common.h"
#include "DeviceTable.h"
#include "Types.h"
#include "xxHash.h"

__global__ void randomizeArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 acc = xxhash(array[id], n);
    acc = xxhash(acc, threadIdx.x);
    acc = xxhash(acc, blockIdx.x);
    acc = xxhash(acc, blockDim.x);
    array[id] = (u32)(acc);
  }
}

__global__ void printArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    printf("%08x: %x\n", id, array[id]);
  }
}

__global__ void setEmpty(u32 *val, u32 capacity) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < capacity) {
    val[id] = empty;
  }
}

__global__ void batchedInsert(DeviceTable *t, u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = array[id];

    for (u32 i = 0; i < t->threshold && v != empty; i += 1) {
      u32 b = i % t->dim;
      u32 key = xxhash(t->seed[b], v) % t->len;
      v = atomicExch(&t->val[b * t->len + key], v);
    }

    // Record number of collisions
    if (v != empty) {
      atomicAdd(&t->collision, 1);
    }
  }
}

__global__ void batchedLookup(DeviceTable *t, u32 *keys, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = keys[id];

    for (u32 i = 0; i < t->dim; i += 1) {
      u32 key = xxhash(t->seed[i], v) % t->len;
      if (t->val[i * t->len + key] == v) {
        break;
      }
    }
  }
}

void wrapper() {
  u32 numEntries = 1 << 24;
  u32 numThreads = 1024;
  u32 entryBlocks = numEntries / numThreads;

  auto t = new DeviceTable(1 << 25, numEntries);

  u32 *array;
  hipMallocManaged(&array, sizeof(u32) * numEntries);
  randomizeArray<<<entryBlocks, numThreads>>>(array, numEntries);

  batchedInsert<<<entryBlocks, numThreads>>>(t, array, numEntries);
  syncCheck();
  while (t->collision > 0) {
    t->reset();
    batchedInsert<<<entryBlocks, numThreads>>>(t, array, numEntries);
    syncCheck();
  }
  batchedLookup<<<entryBlocks, numThreads>>>(t, array, numEntries);
  syncCheck();

  printf("Total number of collisions: %u\n", t->collision);
  syncCheck();

  hipFree(array);
  delete t;

  syncCheck();
}
