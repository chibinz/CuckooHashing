#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"

#include "Common.h"
#include "DeviceTable.h"
#include "Types.h"
#include "xxHash.h"

constexpr u32 empty = (u32)(-1);

__global__ void hello() {
  printf("blockIdx.x=%d/%d blocks, threadIdx.x=%d/%d threads\n", blockIdx.x,
         gridDim.x, threadIdx.x, blockDim.x);
}

__global__ void randomizeArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 acc = xxhash(array[id], n);
    acc = xxhash(acc, threadIdx.x);
    acc = xxhash(acc, blockIdx.x);
    acc = xxhash(acc, blockDim.x);
    array[id] = (u32)(acc);
  }
}

__global__ void printArray(u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    printf("%08x: %x\n", id, array[id]);
  }
}

__global__ void setEmpty(u32 *val, u32 capacity) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < capacity) {
    val[id] = empty;
  }
}

__global__ void batchedInsert(DeviceTable *t, u32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = array[id];

    for (u32 i = 0; i < t->threshold && v != empty; i += 1) {
      u32 b = i % t->dim;
      u32 key = xxhash(t->seed[b], v) % t->len;
      v = atomicExch(&t->val[b * t->len + key], v);
    }

    // Record number of collisions
    if (v != empty) {
      atomicAdd(&t->collision, 1);
    }
  }
}

__global__ void batchedLookup(DeviceTable *t, u32 *keys, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 v = keys[id];

    for (u32 i = 0; i < t->dim; i += 1) {
      u32 key = xxhash(t->seed[i], v) % t->len;
      if (t->val[i * t->len + key] == v) {
        break;
      }
    }
  }
}

void syncCheck() {
  hipDeviceSynchronize();
  auto err = hipGetLastError(); // Get error code
  if (err != hipSuccess) {
    printf("Error: %s!\n", hipGetErrorString(err));
    exit(-1);
  }
}

void tableInit(DeviceTable *t, u32 dim, u32 len) {
  t->dim = dim;
  t->len = len;
  t->threshold = 4 * bit_width(dim * len);
  t->collision = 0;

  u32 numThreads = 256;
  u32 numBlocks = dim * len / numThreads;

  setEmpty<<<numBlocks, numThreads>>>(t->val, dim * len);
  syncCheck();
  randomizeArray<<<1, dim>>>(t->seed, dim);
  syncCheck();
}

DeviceTable *tableNew(u32 dim, u32 len) {
  DeviceTable *t;
  hipMallocManaged(&t, sizeof(DeviceTable));
  hipMallocManaged(&t->val, sizeof(u32) * dim * len);
  hipMallocManaged(&t->seed, sizeof(u32) * dim);

  tableInit(t, dim, len);

  return t;
}

void tableFree(DeviceTable *t) {
  hipFree(t->val);
  hipFree(t->seed);
  hipFree(t);
}

void wrapper() {
  u32 dim = 2;
  u32 len = 1 << 24;
  u32 numEntries = 1 << 24;
  u32 numThreads = 1024;
  u32 entryBlocks = numEntries / numThreads;

  DeviceTable *t = tableNew(dim, len);

  u32 *array;
  hipMallocManaged(&array, sizeof(u32) * numEntries);

  randomizeArray<<<entryBlocks, numThreads>>>(array, numEntries);
  syncCheck();
  batchedInsert<<<entryBlocks, numThreads>>>(t, array, numEntries);
  syncCheck();
  while (t->collision > 0) {
    tableInit(t, dim, len);
    batchedInsert<<<entryBlocks, numThreads>>>(t, array, numEntries);
    syncCheck();
  }
  batchedLookup<<<entryBlocks, numThreads>>>(t, array, numEntries);
  syncCheck();

  printf("Total number of collisions: %u\n", t->collision);
  syncCheck();

  hipFree(array);
  tableFree(t);

  syncCheck();
}
