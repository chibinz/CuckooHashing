#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"
#include "hiprand.h"

#include "DeviceTable.h"
#include "Types.h"
#include "xxHash.h"

constexpr i32 empty = INT32_MIN;

__global__ void hello() {
  printf("blockIdx.x=%d/%d blocks, threadIdx.x=%d/%d threads\n", blockIdx.x,
         gridDim.x, threadIdx.x, blockDim.x);
}

__global__ void genRandArray(i32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id < n) {
    u32 acc = xxhash((u32)(usize)(array), n);
    acc = xxhash(acc, threadIdx.x);
    acc = xxhash(acc, blockIdx.x);
    acc = xxhash(acc, blockDim.x);
    array[id] = (i32)(acc);
  }
}

__global__ void printArray(i32 *array, u32 n) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    printf("%08x: %x\n", id, array[id]);
  }
}

__global__ void tableInit(i32 *table, u32 capacity) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id == 0) {
    table[0] = 0;
  } else if (id < capacity) {
    table[id] = empty;
  }
}

__global__ void batchedInsert(i32 *array, u32 n, i32 *table, u32 capacity) {
  u32 id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    u32 key = xxhash(0, array[id]) % capacity;
    i32 old = atomicCAS(&table[key], empty, array[id]);
    if (old != empty) {
      atomicAdd(&table[0], 1);
    }
  }
}

void syncCheck() {
  hipDeviceSynchronize();
  auto err = hipGetLastError(); // Get error code
  if (err != hipSuccess) {
    printf("Error: %s!\n", hipGetErrorString(err));
    exit(-1);
  }
}

void wrapper() {
  u32 capacity = 2048;
  u32 numEntries = 1024;

  u32 numThreads = 64;
  u32 numBlocks = numEntries / numThreads;
  u32 tableBlocks = capacity / numThreads;

  i32 *array, *table;
  hipMallocManaged(&array, sizeof(u32) * numEntries);
  hipMallocManaged(&table, sizeof(u32) * capacity);

  genRandArray<<<numBlocks, numThreads>>>(array, numEntries);
  tableInit<<<tableBlocks, numThreads>>>(table, capacity);
  batchedInsert<<<1, numThreads>>>(array, numEntries, table, capacity);
  printArray<<<1, numThreads>>>(table, 256);

  hipFree(array);
  hipFree(table);

  syncCheck();
}
